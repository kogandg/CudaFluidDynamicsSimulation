#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void arrayAddition(int* c, const int* a, const int* b, const int size)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

void addWithCuda(int* c, int* a, int* b, int size)
{
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    int NUM_THREADS = 1024;
    int NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;
    arrayAddition << <NUM_BLOCKS, NUM_THREADS >> > (dev_c, dev_a, dev_b, size);

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}